
#include <hip/hip_runtime.h>
extern "C"
__global__ void pocl_add_i32(const int* __restrict x, const int* __restrict y, int* __restrict out)
{
      size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
      out[tid] = x[tid] + y[tid];
}

extern "C"
__global__ void pocl_mul_i32(const int* __restrict x, const int* __restrict y, int* __restrict out)
{
      size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
      out[tid] = x[tid] * y[tid];
}

extern "C"
__global__ void pocl_dnn_conv2d_int8_relu(
  char* __restrict placeholder,
  char* __restrict placeholder1,
  char* __restrict compute,
  int* __restrict placeholder2,
  int* __restrict placeholder3,
  int* __restrict placeholder4,
  int* __restrict placeholder5,
  int* __restrict placeholder6
)
{
      size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
      placeholder6[tid] = placeholder2[tid] * placeholder3[tid];
      placeholder5[tid] = placeholder4[tid] * placeholder2[tid];
}
